#include "common.h"

/** 
 * @brief sample from a uniform distribution
 */
float nn::generateUniformRandom(float range)
{
    return range - float(rand()) / float(RAND_MAX);
}

void nn::apply_grad(float* param, float* grad, float lr, int param_fdim)
{
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float nlr = -lr;

    stat = hipblasSaxpy(handle, param_fdim, &nlr, grad, 1, param, 1);
    if(stat != HIPBLAS_STATUS_SUCCESS)
    {
        printf("hipblasSaxpy failed\n");
    }
}
