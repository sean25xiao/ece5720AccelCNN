#include "input.h"

Input::Input(nn::shape_3d input_shape)
{
    this->type = INPUT_LAYER_TYPE;
    this->input_shape = input_shape;
    this->output_shape = input_shape;

    this->output_fdim = output_shape.width*output_shape.height*output_shape.channel;

    printf("+ Input layer,  output_shape=(%d,%d,%d)\n",
            output_shape.width, output_shape.height, output_shape.channel);
    hipMalloc(&output, sizeof(float) * output_fdim);
    hipMalloc(&d_output, sizeof(float) * output_fdim);
}

Input::~Input()
{
    hipFree(output);
    hipFree(d_output);
}

void Input::feed(float* sample)
{
    hipError_t cudaStatus; 
    cudaStatus = hipMemcpy(output, sample, sizeof(float) * output_fdim, hipMemcpyHostToDevice);
    if (hipSuccess != cudaStatus)
    {
        throw "Failed: cannot copy memory in Inputlayer feed\n";
    }
}

void Input::forward(){};
void Input::backward(){};
void Input::clear()
{
    hipMemset(output, 0x00, sizeof(float) * output_fdim);
    hipMemset(d_output, 0x00, sizeof(float) * output_fdim);
}






