#include "hip/hip_runtime.h"
#include "conv2d.h"
#include <cstdio>

Conv2d::Conv2d(int input_width, int input_height, int input_channels, int kernel_size, int output_channels, int stride)
{
    // assign input shape
    this->in_w = input_width;
    this->in_h = input_height;
    this->in_c = input_channels;

    // assign kernel shape and stride
    this->k_sz = kernel_size;
    this->stride = stride;

    // output shape
    this->ot_w = (in_w - kernel_size) / stride + 1;
    this->ot_h = (in_h - kernel_size) / stride + 1;
    this->ot_c = output_channels;

    printf("Conv Layer, input_shape=(%d,%d,%d), output_shape=(%d,%d,%d)\n",in_c, in_w, in_h, ot_c, ot_w, ot_h);

    this->M = k_sz * k_sz;
    this->N = ot_c;
    this->O = ot_w * ot_h * ot_c;
    this->weight_dim = N*M;
    // random initialize weight and bias
    float h_weight[N][M];
    float h_bias[N];

    for (int i = 0; i < N; ++i)
    {
        h_bias[i] = 0.5f - float(rand()) / float(RAND_MAX);
        // h_bias[i] = 0; // DEBUG

        for (int j = 0; j < M; ++j)
        {
            h_weight[i][j] = 0.5f - float(rand()) / float(RAND_MAX);
            // h_weight[i][j] = 1; // DEBUG
        }
    }

    // allocate cuda memory
    hipMalloc(&output, sizeof(float) * O);
    hipMalloc(&weight, sizeof(float) * M * N);
    hipMalloc(&bias, sizeof(float) * N);

    hipMalloc(&d_output, sizeof(float) * O);
    hipMalloc(&d_weight, sizeof(float) * M * N);

    // copy initialized weight and biase from host memory to device memory
    hipMemcpy(bias, h_bias, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(weight, h_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);
}

Conv2d::~Conv2d()
{
    // Free cuda memory
    hipFree(output);
    hipFree(weight);
    hipFree(bias);

    hipFree(d_output);
    hipFree(d_weight);
}

// Memory clear function used between iterations
void Conv2d::forward_reset()
{
    hipMemset(output, 0x00, sizeof(float) * O);
}

void Conv2d::backward_reset()
{
    hipMemset(d_output, 0x00, sizeof(float) * O);
    hipMemset(d_weight, 0x00, sizeof(float) * M * N);
}

/* 
 *   Forward pass of Convolutional Layer, 
 *   IMPORTANT: prev_output should be the output of last layer 
 */
void Conv2d::forward(float *prev_output)
{
    // Convolution
    forward_conv<<<64, 64>>>(prev_output, output, weight, k_sz, ot_w, ot_h, ot_c);

    // Add bias
    forward_add_bias<<<64, 64>>>(output, bias, ot_w, ot_h, ot_c);
};

/*
 *  Backward pass of Convolutional Layer,
 *  IMPORTANT: make sure d_output of current layer is already assigned before run this function
 */
void Conv2d::backward(float *prev_output, float *prev_d_output)
{
    // Compute gradient of weight
    backward_gradient_weight<<<64, 64>>>(d_weight, d_output, prev_output, k_sz, ot_w, ot_h, ot_c);
    // Update Bias
    backward_update_bias<<<64, 64>>>(bias, d_output, ot_w, ot_h, ot_c);
    // Update prev_d_output
    if ( prev_d_output == NULL) return;
    backward_gradient_prev<<<64, 64>>>(prev_d_output, weight, d_output, k_sz, ot_w, ot_h, ot_c);
};

__global__ void backward_gradient_weight(float *d_weight, float *d_output, float *prev_output,
                                         int k_sz, int ot_w, int ot_h, int ot_c)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = ot_c * k_sz * k_sz * ot_w * ot_h;

    float d = pow(24.0f, 2.0f);

    for (int n = N * pos / size; n < N * (pos + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % ot_c);
        int i2 = ((idx /= ot_c) % k_sz);
        int i3 = ((idx /= k_sz) % k_sz);
        int i4 = ((idx /= k_sz) % ot_w);
        int i5 = ((idx /= ot_w) % ot_h);

        atomicAdd(&d_weight[i1 * k_sz * k_sz + i2 * k_sz + i3], d_output[i1 * ot_w * ot_h + i4 * ot_h + i5] * prev_output[(i4 + i2) * (k_sz + ot_h) + i5 + i3] / d);
        // atomicAdd(&d_weight[i1][i2][i3], d_output[i1][i4][i5] * prev_output[i4 + i2][i5 + i3] / d);
    }
}

__global__ void backward_update_bias(float *bias, float *d_output,
                                     int ot_w, int ot_h, int ot_c)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = ot_c * ot_w * ot_h;
    float d = pow(24.0f, 2.0f);

    for (int n = N * pos / size; n < N * (pos + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % ot_c);
        int i2 = ((idx /= ot_c) % ot_w);
        int i3 = ((idx /= ot_w) % ot_h);

        atomicAdd(&bias[i1], LEARNING_RATE * d_output[i1 * ot_w * ot_h + i2 * ot_h + i3] / d);
        // atomicAdd(&bias[i1], LEARNING_RATE * d_output[i1][i2][i3] / d);
    }
}

__global__ void backward_gradient_prev(float *prev_d_output, float *weight, float *d_output,
                                       int k_sz, int ot_w, int ot_h, int ot_c)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = k_sz * k_sz * ot_c * ot_w * ot_h;

    for (int n = N * pos / size; n < N * (pos + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % 1);
        int i2 = ((idx /= 1) % k_sz);
        int i3 = ((idx /= k_sz) % k_sz);
        int i4 = ((idx /= k_sz) % ot_c); // ?
        int i5 = ((idx /= ot_c) % ot_w);
        int i6 = ((idx /= ot_w) % ot_h);

        atomicAdd(&prev_d_output[i4 * (ot_w * 4 + k_sz) + (i5 * 4 + i2) * (ot_h * 4 + k_sz) + i6 * 4 + i3],
                  weight[i1 * k_sz * k_sz + i2 * k_sz + i3] * d_output[i4 * ot_w * ot_h + i5 * ot_h + i6]);
        // atomicAdd(&prev_d_output[i4][i5 * 4 + i2][i6 * 4 + i3], weight[i1][i2][i3] * d_output[i4][i5][i6]);
    }
}

__global__ void forward_conv(float *input, float *output, float *weight,
                             int k_sz, int ot_w, int ot_h, int ot_c)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = k_sz * k_sz * ot_c * ot_w * ot_h;

    for (int n = N * pos / size; n < N * (pos + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % k_sz);
        int i2 = ((idx /= k_sz) % k_sz);
        int i3 = ((idx /= k_sz) % ot_c);
        int i4 = ((idx /= ot_c) % ot_w);
        int i5 = ((idx /= ot_w) % ot_h);

        atomicAdd(&output[i3*ot_w*ot_h+i4*ot_h+i5], weight[i3*k_sz*k_sz+i1*k_sz + i2] * input[(i4 + i1)*(k_sz+ot_h)+i5 + i2]);
        // atomicAdd(&output[i3][i4][i5], weight[i3][i1][i2] * input[i4 + i1][i5 + i2]);
    }
}

__global__ void forward_add_bias(float *output, float *bias, int ot_w, int ot_h, int ot_c)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;

    int N = ot_c * ot_w * ot_h;

    for (int n = N * pos / size; n < N * (pos + 1) / size; ++n)
    {
        int idx = n;
        int i1 = ((idx /= 1) % ot_c);
        int i2 = ((idx /= ot_c) % ot_w);
        int i3 = ((idx /= ot_w) % ot_h);

        output[i1 *ot_w*ot_h+i2*ot_h+i3] += bias[i1];
        // output[i1][i2][i3] += bias[i1];
    }
}
