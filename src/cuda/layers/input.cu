#include "input.h"

Input::Input(int width, int height){
    this->O = width*height;
    hipMalloc(&output, sizeof(float) * width* height);
}

Input::~Input(){
    hipFree(output);
}

void Input::forward(float* data){
    hipMemcpy(output, data, sizeof(float) * O, hipMemcpyHostToDevice);
}

