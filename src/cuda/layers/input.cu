#include "input.h"
#include <cstdio>

Input::Input(int width, int height){
    this->O = width*height;
    hipMalloc(&output, sizeof(float) * width* height);
    printf("Input Layer, output_shape=(%d,%d)\n", width, height);
}

Input::~Input(){
    hipFree(output);
}

void Input::forward(float* data){
    hipMemcpy(output, data, sizeof(float) * O, hipMemcpyHostToDevice);
}

