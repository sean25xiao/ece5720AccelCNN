/*
 *  CUDA for Convolutional Neural Network 
 *  
 *  by Zikun Xu, Bangqi Xiao
 */
 #define USE_MNIST_LOADER
 #define MNIST_DOUBLE
 #include "mnist.h"

 #include <hip/hip_runtime.h>
 #include <cstdio>
 #include <time.h>

 mnist_data *train_set, *test_set; // pointer to dataset struct
 int train_cnt, test_cnt; // sample count

 void load_minist()
 {
	mnist_load("../data/train-images.idx3-ubyte", "../data/train-labels.idx1-ubyte",
		&train_set, &train_cnt);
	mnist_load("../data/t10k-images.idx3-ubyte", "../data/t10k-labels.idx1-ubyte",
		&test_set, &test_cnt);
 }

 int main(int argc, const char ** argv)
 {
     // CUDA initialisation
     hipError_t err = NULL;
     err = hipInit(0);
     if( hipSuccess != err ){
         fprintf(stderr, "CUDA initialisation failed with error code - %d\n", err);
         return 1;
     }
     // loading dataset
     load_minist();

     build_model();
     train();
     test();

     return 0;
 }